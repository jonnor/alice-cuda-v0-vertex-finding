#include "hip/hip_runtime.h"
/* This file is part of a project that implements GPU based 
 * v0 vertex finding for use with AliROOT in ALICE
 * http://ri-pro.hive.no/prosjekter/EN2010-01/ 
 * Code at http://gitorious.org/cuda-alice-vertex-finding
 */

/**************************************************************************
 * Copyright(c) 2010  Vestfold University College, All rights reserved.   *
 *                                                                        *
 * Authors: Jon Nordby, Lars Bratrud                                      *
 *                                                                        *
 * Permission to use, copy, modify and distribute this software and its   *
 * documentation strictly for non-commercial purposes is hereby granted   *
 * without fee, provided that the above copyright notice appears in all   *
 * copies and that both the copyright notice and this permission notice   *
 * appear in the supporting documentation. The authors make no claims     *
 * about the suitability of this software for any purpose. It is          *
 * provided "as is" without express or implied warranty.                  *
 **************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>

#include "AliCudaDefs.h"
#include "AliCudaV0vertexer.h"
#include "aliexternaltrackparam.cu"
#include "aliv0vertexer.cu"

const int TRACK_SIZE = sizeof(struct trackparam);
const int HELIX_SIZE = sizeof(Double_t)*6;
const int VERTEX_SIZE = sizeof(struct privertex);

Int_t cuda_v0_vertexer(struct privertex* vtx, struct trackparam* tracks, 
                        Int_t ntrks, Double_t b) {

    if (ntrks < 2) return 0;

    // Host data
    Int_t nv0s=0;

    // Declare and allocate device data
    struct trackparam* tracks_d;
    struct privertex* vtx_d;
    hipMalloc((void**)&vtx_d, VERTEX_SIZE);
    hipMalloc((void**)&tracks_d, TRACK_SIZE*ntrks);

    Int_t *nv0s_d;
    hipMalloc((void**)&nv0s_d, sizeof(Int_t));

    Int_t *ptracks_d;
    Int_t *ntracks_d;
    hipMalloc((void**)&ntracks_d, sizeof(Int_t)*ntrks);
    hipMalloc((void**)&ptracks_d, sizeof(Int_t)*ntrks);

    Int_t *npos_d; Int_t *nneg_d;
    hipMalloc((void**)&npos_d, sizeof(Int_t));
    hipMalloc((void**)&nneg_d, sizeof(Int_t));
    printf("Malloc DONE\n");

    // Copy data to device
    hipMemcpy(tracks_d, tracks, TRACK_SIZE*ntrks, hipMemcpyHostToDevice);
    hipMemcpy(vtx_d, vtx, VERTEX_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(nv0s_d, &nv0s, sizeof(Int_t), hipMemcpyHostToDevice);
    printf("Memcpy HostToDevice DONE\n");

    // Execute
    const Int_t warpsize = 32;
    dim3  grid(ceil(ntrks/warpsize), 1, 1); // x,y,z
    dim3  block(warpsize, 1, 1); // x,y,z

    SortTracks_kernel<<<grid, block>>>(vtx_d, tracks_d, 
                                        ptracks_d, ntracks_d, 
                                        npos_d, nneg_d,
                                        ntrks, b);
    hipDeviceSynchronize();
    printf("SortTracks kernel execution DONE\n");


    // DEBUG
    Int_t npos, nneg;
    hipMemcpy(&npos, npos_d, sizeof(Int_t), hipMemcpyDeviceToHost);
    hipMemcpy(&nneg, nneg_d, sizeof(Int_t), hipMemcpyDeviceToHost);
    printf("SortTracks: npos=%d, nneg=%d\n", npos, nneg);

    Tracks2V0vertices_kernel<<<grid, block>>>(vtx_d, tracks_d,
                                                ptracks_d, ntracks_d, 
                                                npos_d, nneg_d,
                                                nv0s_d, b);
    hipDeviceSynchronize();
    printf("Tracks2V0vertices kernel execution DONE\n");

    // Copy data back and clean up
    hipMemcpy(vtx, vtx_d, VERTEX_SIZE, hipMemcpyDeviceToHost);
    hipFree(vtx_d); hipFree(tracks_d);

    hipMemcpy(&nv0s, nv0s_d, sizeof(Int_t), hipMemcpyDeviceToHost);
    hipFree(nv0s_d);

    return nv0s;
}

int test_cuda_v0_vertexer()
{

    Double_t b = -5.00668;
    struct trackparam *tp;
    Double_t *hp;

    // Allocate memory
    tp = (struct trackparam*)malloc(TRACK_SIZE);
    hp = (Double_t *)malloc(HELIX_SIZE);

    // Initialize data
    tp->fP[0] = -0.00429969;
    tp->fP[1] = -4.56162;
    tp->fP[2] = 2.38928e-09;
    tp->fP[3] = 0.855637;
    tp->fP[4] = -1.96397;
    tp->fAlpha = 1.90909;
    tp->fX = -0.00462971;
    for(int i=0; i<6;i++) hp[i] = 0;

    printf("GetHelixParameters\n");
    GetHelixParameters(tp, hp, b);
    for (int i=0; i<6; i++) printf("%f\n", hp[i]);

    // TODO: find real inputdata and test
    Double_t t = 3; 
    Double_t rv[3], d[3], dd[3];
    Evaluate(hp, t, rv, d, dd);

    // TODO: find real inputdata and test
    Double_t xk = 50.0;
    PropagateTo(tp, xk, b);
    printf("PropagateTo\n");
    for (int i=0; i<5; i++) printf("%f\n",tp->fP[i]);

    // TODO: find real inputdata and test
    Double_t xv = 1.0, yv = 1.0;
    Double_t d_lin = GetLinearD(tp, xv, yv);
    printf("GetLinearD = %f\n", d_lin);
    Double_t d_ = GetD(tp, xv, yv, b);
    printf("GetD = %f\n", d_);


    // TODO: find real inputdata and test. Must include covariance matrix
    struct trackparam *tp2;
    tp2 = (struct trackparam*)malloc(TRACK_SIZE);

    tp2->fP[0] = -0.0315877;
    tp2->fP[1] = -4.54952;
    tp2->fP[2] = 3.74249e-09;
    tp2->fP[3] = 1.15249;
    tp2->fP[4] = 1.67247;
    tp2->fAlpha = 0.107172;
    tp2->fX = 0.000891429;

    Double_t xp=1.0, xn=1.0;
    Double_t dca = 1.0;
    dca = GetDCA(tp, tp2, b, xn, xp);
    printf("GetDCA = %f\n", dca);

    // v0 vertexer
    struct privertex *vtxT3d;
    vtxT3d = (struct privertex *) malloc(sizeof(struct privertex));
    vtxT3d->fPosition[0] = 0.0;
    vtxT3d->fPosition[1] = 0.0;
    vtxT3d->fPosition[2] = 0.0;

    const int NTRACKS=2;
    struct trackparam *tracks;
    tracks = (struct trackparam*)malloc(sizeof(struct trackparam)*NTRACKS);
    tracks[0] = *tp;
    tracks[1] = *tp2;

    printf("Tracks2V0vertices\n");
//    Tracks2V0vertices(vtxT3d, tracks, NTRACKS, b);
//     Tracks2V0vertices_kernel<<<1,1>>>(vtxT3d, tracks, NTRACKS, b);
//     hipDeviceSynchronize();

    cuda_v0_vertexer(vtxT3d, tracks, NTRACKS, b);

    // Cleanup
    free(tp); free(hp); free(tp2); free(vtxT3d);

    return 1;
}

int main() {
    return test_cuda_v0_vertexer();
}
